#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <string.h>
#include <malloc.h>
#include <stdint.h>

 struct pix {
    uint8_t R = 0;
    uint8_t G = 0;
    uint8_t B = 0;
};
 struct BMPHeader {
    uint8_t n1;//1
    uint8_t n2;//1
    uint32_t Size;//4
    uint16_t Reserved1;//2
    uint16_t Reserved2;//2
    uint32_t OffsetBits;//4
    uint32_t Size2;//4
    uint32_t Width;//4
    uint32_t Height;//4
    uint16_t Planes;//2
    uint16_t BytePerPix;//2
    uint32_t Compression;//4
    uint32_t SizeImage;//4
    uint32_t XpelsPerMeter;//4
    uint32_t YpelsPerMeter;//4
    uint32_t ColorsUsed;//4
    uint32_t ColorsImportant;//4
};

__global__ void addminor(pix* img, int32_t* height, int32_t* width)
{
    int sumR = 0;
    int sumG = 0;
    int sumB = 0;
    int count = 0;
    int ii = blockIdx.x;//текущий пиксель высота
    int jj = threadIdx.x;//текущий пиксель ширина
    for (int i = ii - 1; i <= ii + 1; i++) {//высота
        for (int j = jj - 1; j <= jj + 1; j++) {//ширина
            if (i >= 0 && i < *height && j >= 0 && j < *width) {//не выходим за границы массива
                sumR += img[i * *width + j].R;
                sumG += img[i * *width + j].G;
                sumB += img[i * *width + j].B;
                count++;
            }
        }
    }
    sumR /= count;
    sumG /= count;
    sumB /= count;
    img[ii * *width + jj].R = sumR;
    img[ii * *width + jj].G = sumG;
    img[ii * *width + jj].B = sumB;
}


__host__ int StartCuda(pix* img, int32_t height, int32_t width, int32_t BytePerPix) {
    int32_t* dev_width;
    hipMalloc((void**)&dev_width, sizeof(int32_t));
    hipMemcpy(dev_width, &width, sizeof(int32_t), hipMemcpyHostToDevice);

    int32_t* dev_height;
    hipMalloc((void**)&dev_height, sizeof(int32_t));
    hipMemcpy(dev_height, &height, sizeof(int32_t), hipMemcpyHostToDevice);

    pix* dev_img;
    hipMalloc((void**)&dev_img, width * height * BytePerPix);
    hipMemcpy(dev_img, img, width * height * BytePerPix, hipMemcpyHostToDevice);

    addminor <<< height, width >>> (dev_img, dev_height, dev_width);

    hipMemcpy(img, dev_img, width * height * BytePerPix, hipMemcpyDeviceToHost);

    hipFree(dev_width);
    hipFree(dev_height);
    hipFree(dev_img);
    
    return 0;
}

__host__ int MakeBMP(pix* img, FILE* file, int32_t width, int32_t height, int32_t BytePerPix)
{
    struct BMPHeader BmpH;
    fseek(file, SEEK_SET, SEEK_SET);
    fread(&BmpH, sizeof(BMPHeader), 1, file);

    FILE* file2;
    file2 = fopen("1mod.bmp", "wb");
    fwrite(&BmpH, sizeof(BMPHeader), 1, file2);


    int LineLenght = (width * BytePerPix + BytePerPix) / 4 * 4;
    uint8_t* arr = (uint8_t*)calloc(height * LineLenght, 1);
    for (int i = 0, int ii = 0; i < width * height; i++) {
        if (i % width  == 0 && i != 0) {
            for (int j = 0; j < (LineLenght - width * BytePerPix); j++) {//здесь дополнял строку нулями
                arr[ii] = 0;
                ii++;
            }
        }
        
        arr[ii] = img[i].R;
        ii++;
        arr[ii] = img[i].G;
        ii++;
        arr[ii] = img[i].B;
        ii++;
        //printf("\n current ii is %d ", ii);
    }
    fseek(file2, 54, SEEK_SET);
    printf("\n");
    for (int i = 0; i < height * LineLenght; i++) {
        //printf("%3d ", arr[i]);
        
        fwrite(&arr[i], 1, sizeof(arr[i]), file2);
    }

    fclose(file2);
    free(arr);
    return 0;
}

__host__ int main()
{
    FILE* file;
    file = fopen("1.bmp", "rb");

    //check that the BMP file
    char name1, name2;//B M
    fread(&name1, 1, 1, file);
    fread(&name2, 1, 1, file);
    if (name1 != 'B' && name2 != 'm') {
        printf("Incorrect file format\n");
        return 0;
    }

    //looking at the image dimensions
    int32_t width; // Ширина изображения (4 байта)
    fseek(file, 18, SEEK_SET);
    fread(&width, 4, 1, file);

    int32_t height; // Высота изображения (4 байта)
    fseek(file, 22, SEEK_SET);
    fread(&height, 4, 1, file);

    int32_t BytePerPix;
    fseek(file, 28, SEEK_SET);
    fread(&BytePerPix, 4, 1, file);
    BytePerPix = BytePerPix / 8;

    int LineLenght = (width * BytePerPix + BytePerPix) / 4 * 4;

    //ссчитываем изображение, на каждый пиксель будет уходить BitPerPix / 8 байт.
    struct pix* img = (pix*)calloc(width * height, 3);//new pix [width * height * BytePerPix + width * BytePerPix % 4];
    //в бмп есть выравнивание, если строка пикселей состоит не из кратного 4 числа пикселей то её дополняют 0
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            fseek(file, 54 + i * LineLenght + j * BytePerPix, SEEK_SET);
            fread(&img[i * width + j].R, 1, 1, file);
            fseek(file, 54 + i * LineLenght + j * BytePerPix + 1, SEEK_SET);
            fread(&img[i * width + j].G, 1, 1, file);
            fseek(file, 54 + i * LineLenght + j * BytePerPix + 2, SEEK_SET);
            fread(&img[i * width + j].B, 1, 1, file);
        }
    }
    
    //printf("LineLenght %d , width %d , height %d , BytePerPix %d\n", LineLenght, width, height, BytePerPix);
    
    StartCuda(img, height, width, BytePerPix);

    MakeBMP(img, file, width, height, BytePerPix);

    free(img);
    fclose(file);
    return 0;
}